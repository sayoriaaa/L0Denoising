#include "hip/hip_runtime.h"
#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>
#include <hipfft/hipfft.h>
#include <complex>

#include <hip/hip_runtime.h>
#include <>
#include <cmath>

namespace py = pybind11;

py::array_t<std::complex<float>> cufft_fft2(py::array_t<std::complex<float>> input) {
    py::buffer_info buf_info = input.request();
    std::complex<float>* data_ptr = static_cast<std::complex<float>*>(buf_info.ptr);

    int height = buf_info.shape[0];
    int width = buf_info.shape[1];

    hipfftHandle fft_plan;
    hipfftPlan2d(&fft_plan, height, width, HIPFFT_C2C);

    hipfftComplex* d_in;
    hipMalloc((void**)&d_in, sizeof(hipfftComplex) * height * width);
    hipMemcpy(d_in, data_ptr, sizeof(std::complex<float>) * height * width, hipMemcpyHostToDevice);

    hipfftComplex* d_out;
    hipMalloc((void**)&d_out, sizeof(hipfftComplex) * height * width);

    hipfftExecC2C(fft_plan, d_in, d_out, HIPFFT_FORWARD);

    hipfftDestroy(fft_plan);
    hipFree(d_in);

    // Create a NumPy array from d_out data and return
    py::array_t<std::complex<float>> result({height, width});
    py::buffer_info buf_info_result = result.request();
    std::complex<float>* result_ptr = static_cast<std::complex<float>*>(buf_info_result.ptr);
    hipMemcpy(result_ptr, d_out, sizeof(hipfftComplex) * height * width, hipMemcpyDeviceToHost);

    hipFree(d_out);

    return result;
}

__global__ void normalizeIfftResult(hipfftComplex* d_out, float normalization_factor, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        d_out[idx].x *= normalization_factor;
        d_out[idx].y *= normalization_factor;
    }
}

py::array_t<std::complex<float>> cufft_ifft2(py::array_t<std::complex<float>> input) {
    py::buffer_info buf_info = input.request();
    std::complex<float>* data_ptr = static_cast<std::complex<float>*>(buf_info.ptr);

    int height = buf_info.shape[0];
    int width = buf_info.shape[1];

    hipfftHandle ifft_plan;
    hipfftPlan2d(&ifft_plan, height, width, HIPFFT_C2C);

    hipfftComplex* d_in;
    hipMalloc((void**)&d_in, sizeof(hipfftComplex) * height * width);
    hipMemcpy(d_in, data_ptr, sizeof(std::complex<float>) * height * width, hipMemcpyHostToDevice);

    hipfftComplex* d_out;
    hipMalloc((void**)&d_out, sizeof(hipfftComplex) * height * width);

    hipfftExecC2C(ifft_plan, d_in, d_out, HIPFFT_BACKWARD);

    hipfftDestroy(ifft_plan);
    hipFree(d_in);

    // Normalize the IFFT result
    float normalization_factor = 1.0f / (height * width);
    normalizeIfftResult<<<(height * width + 255) / 256, 256>>>(d_out, normalization_factor, height * width);

    // Create a NumPy array from d_out data and return
    py::array_t<std::complex<float>> result({height, width});
    py::buffer_info buf_info_result = result.request();
    std::complex<float>* result_ptr = static_cast<std::complex<float>*>(buf_info_result.ptr);
    hipMemcpy(result_ptr, d_out, sizeof(hipfftComplex) * height * width, hipMemcpyDeviceToHost);

    hipFree(d_out);

    return result;
}

__global__ void computeHV(float* H, float* V, int height, int width, float lambda, float beta) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < width - 1 && row < height - 1) {
        int index = row * width + col;
        int index_yp = (row + 1) * width + col;
        int index_xp = row * width + (col + 1);

        float point = H[index];
        float point_xp = H[index_xp];
        float point_yp = H[index_yp];

        float dy = point_yp - point;
        float dx = point_xp - point;

        float norm = std::pow(dy, 2) + std::pow(dx, 2);
        if (norm < lambda / beta) {
            V[index] = 0.;
            H[index] = 0.;
        }
        else{
            V[index] = dy;
            H[index] = dx;
        }
    }
    else if(col == width-1 || row == height-1) {
        int index = row * width + col;
        V[index] = 0.;
        H[index] = 0.;
    }
}

py::tuple updateHV(const py::array_t<float>& S, int height, int width, float lambda, float beta) {
    py::buffer_info buf_info = S.request();
    const float* S_ptr = static_cast<float*>(buf_info.ptr);

    int chan = buf_info.shape[0];

    py::array_t<float> Hs = py::array_t<float>({chan, height, width});
    py::array_t<float> Vs = py::array_t<float>({chan, height, width});

    float* Hs_ptr = static_cast<float*>(Hs.request().ptr);
    float* Vs_ptr = static_cast<float*>(Vs.request().ptr);

    for (int i = 0; i < chan; ++i) {
        float* d_H;
        float* d_V;

        hipMalloc(&d_H, height * width * sizeof(float));
        hipMalloc(&d_V, height * width * sizeof(float));

        dim3 blockDim(16, 16);
        dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);

        hipMemcpy(d_H, S_ptr + (i * height * width), height * width * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_V, S_ptr + (i * height * width), height * width * sizeof(float), hipMemcpyHostToDevice);

        computeHV<<<gridDim, blockDim>>>(d_H, d_V, height, width, lambda, beta);

        hipDeviceSynchronize();

        hipMemcpy(Hs_ptr + (i * height * width), d_H, height * width * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(Vs_ptr + (i * height * width), d_V, height * width * sizeof(float), hipMemcpyDeviceToHost);

        hipFree(d_H);
        hipFree(d_V);
    }

    return py::make_tuple(Hs, Vs);
}



PYBIND11_MODULE(cuda_utils, m) {
    m.def("fft2", &cufft_fft2, "Compute FFT using CUFFT");
    m.def("ifft2", &cufft_ifft2, "Compute IFFT using CUFFT");
    m.def("updateHV", &updateHV, "Update H and V using CUDA");
}