#include "hip/hip_runtime.h"
#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>
#include <hipfft/hipfft.h>
#include <complex>

namespace py = pybind11;

py::array_t<std::complex<float>> cufft_fft2(py::array_t<std::complex<float>> input) {
    py::buffer_info buf_info = input.request();
    std::complex<float>* data_ptr = static_cast<std::complex<float>*>(buf_info.ptr);

    int height = buf_info.shape[0];
    int width = buf_info.shape[1];

    hipfftHandle fft_plan;
    hipfftPlan2d(&fft_plan, height, width, HIPFFT_C2C);

    hipfftComplex* d_in;
    hipMalloc((void**)&d_in, sizeof(hipfftComplex) * height * width);
    hipMemcpy(d_in, data_ptr, sizeof(std::complex<float>) * height * width, hipMemcpyHostToDevice);

    hipfftComplex* d_out;
    hipMalloc((void**)&d_out, sizeof(hipfftComplex) * height * width);

    hipfftExecC2C(fft_plan, d_in, d_out, HIPFFT_FORWARD);

    hipfftDestroy(fft_plan);
    hipFree(d_in);

    // Create a NumPy array from d_out data and return
    py::array_t<std::complex<float>> result({height, width});
    py::buffer_info buf_info_result = result.request();
    std::complex<float>* result_ptr = static_cast<std::complex<float>*>(buf_info_result.ptr);
    hipMemcpy(result_ptr, d_out, sizeof(hipfftComplex) * height * width, hipMemcpyDeviceToHost);

    hipFree(d_out);

    return result;
}

__global__ void normalizeIfftResult(hipfftComplex* d_out, float normalization_factor, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        d_out[idx].x *= normalization_factor;
        d_out[idx].y *= normalization_factor;
    }
}

py::array_t<std::complex<float>> cufft_ifft2(py::array_t<std::complex<float>> input) {
    py::buffer_info buf_info = input.request();
    std::complex<float>* data_ptr = static_cast<std::complex<float>*>(buf_info.ptr);

    int height = buf_info.shape[0];
    int width = buf_info.shape[1];

    hipfftHandle ifft_plan;
    hipfftPlan2d(&ifft_plan, height, width, HIPFFT_C2C);

    hipfftComplex* d_in;
    hipMalloc((void**)&d_in, sizeof(hipfftComplex) * height * width);
    hipMemcpy(d_in, data_ptr, sizeof(std::complex<float>) * height * width, hipMemcpyHostToDevice);

    hipfftComplex* d_out;
    hipMalloc((void**)&d_out, sizeof(hipfftComplex) * height * width);

    hipfftExecC2C(ifft_plan, d_in, d_out, HIPFFT_BACKWARD);

    hipfftDestroy(ifft_plan);
    hipFree(d_in);

    // Normalize the IFFT result
    float normalization_factor = 1.0f / (height * width);
    normalizeIfftResult<<<(height * width + 255) / 256, 256>>>(d_out, normalization_factor, height * width);

    // Create a NumPy array from d_out data and return
    py::array_t<std::complex<float>> result({height, width});
    py::buffer_info buf_info_result = result.request();
    std::complex<float>* result_ptr = static_cast<std::complex<float>*>(buf_info_result.ptr);
    hipMemcpy(result_ptr, d_out, sizeof(hipfftComplex) * height * width, hipMemcpyDeviceToHost);

    hipFree(d_out);

    return result;
}

PYBIND11_MODULE(cuda_utils, m) {
    m.def("fft2", &cufft_fft2, "Compute FFT using CUFFT");
    m.def("ifft2", &cufft_ifft2, "Compute IFFT using CUFFT");
}